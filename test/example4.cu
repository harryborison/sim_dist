
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>
__global__ void addKernel(int *c , const int *a, const int *b){
	int i = threadIdx.x;
	c[i] = a[i]+ b[i];
}
int main(void)
{
	int a[100000];
	int b[100000];
	int *ary1=0;
	int *ary2=0;
	int *ary3=0;
	for(int i=0;i<100000;i++)
	{
		a[i] = i;
	}
	hipMalloc((void**)&ary1 , 100000*sizeof(int));
	hipMalloc((void**)&ary2 , 100000*sizeof(int));
	hipMalloc((void**)&ary3 , 100000*sizeof(int));
	
	hipMemcpy(ary2, a, 100000*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(ary1, a, 100000*sizeof(int),hipMemcpyHostToDevice);
	/*

	for(int i=0;i<100000;i++)
	{
		ary3[i] = ary1[i] + ary2[i];
	}
	*/
	addKernel<<<1,5000>>>(ary3,ary1,ary2);
	hipMemcpy(b, ary3 ,100000*sizeof(int),hipMemcpyDeviceToHost);
	for(int i =0; i < 100000; i++)
	{
		if(i/500 ==0)
		{
			printf("%d ",b[i]);
		}
	}
	hipFree(ary1);
	hipFree(ary2);
	hipFree(ary3);
	return 0;


}
