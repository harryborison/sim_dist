
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdio>



__global__ void input(  int *output)
{
	__shared__ int s_data[1024];
	for(int i= 0 ; i < 1024 ; i++)
	{
		s_data[i] = 2;
	}

	__syncthreads();
	/*
	for(int i=0 ; i < 32; i++)
	{
	int t = threadIdx.x + i *32;
	output[t]=s_data[t]; 	

	}*/
	for(int i=0; i < 32 ; i++)
	{
	output[threadIdx.x*32+i] = s_data[threadIdx.x*32+i];

	}

}



int main(void)
{

	int *ary;
	hipMalloc((void**)&ary, 1024*sizeof(int));
	input<<<1,32>>>(ary);
	int *ary2;
	ary2= (int *)malloc(sizeof(int)*1024);
	hipMemcpy(ary2,ary,sizeof(int)*1024,hipMemcpyDeviceToHost);

	printf("final result : %d %d %d",ary2[0],ary2[1],ary2[1023]);







	return 0;
}
