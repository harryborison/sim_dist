
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>

int main(void)
{
	int a[100000];
	int b[100000];
	int *ary1;
	int *ary2;
	int *ary3;
	for(int i=0;i<100000;i++)
	{
		a[i] = i;
	}
	hipMalloc((void**)&ary1 , 100000*sizeof(int));
	hipMalloc((void**)&ary2 , 100000*sizeof(int));
	hipMalloc((void**)&ary3 , 100000*sizeof(int));
	
	hipMemcpy(ary2, a, 100000*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(ary1, a, 100000*sizeof(int),hipMemcpyHostToDevice);
	

	for(int i=0;i<100000;i++)
	{
		ary3[i] = ary1[i] + ary2[i];
	}
	hipMemcpy(b, ary3 ,100000*sizeof(int),hipMemcpyDeviceToHost);
	for(int i =0; i < 100000; i++)
	{
		if(i/500 ==0)
		{
			printf("%d ",i);
		}
	}
	hipFree(ary1);
	hipFree(ary2);
	hipFree(ary3);
	return 0;


}
