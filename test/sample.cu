
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>
#include <stdlib.h>
#define SIZE 1024*128*512



// int == 4byte
// 1GB 256  1kb
// 256 1024 1mb
// 256 1024 1024 1GB



__global__ void input(int *a, int *b)
{
	int i=blockIdx.x*blockDim.x*512 + threadIdx.x*512;
	int t=i+2048;
	for(;i<t;i++)
	{
	a[i]=b[i];
	}
}
int main(void)
{
	
	int *arr;
	int *arr2;
	int *carr=0;
	int *carr2=0;
	arr= (int *)malloc(sizeof(int)*SIZE);
	arr2= (int *)malloc(sizeof(int)*SIZE);
	for(int i=0; i<SIZE; i++)
	{
	arr[i] = i;

	}

	hipMalloc((void**)&carr2,sizeof(int)*SIZE);
	hipMalloc((void**)&carr,sizeof(int)*SIZE);
	hipMemcpy(carr,arr,sizeof(int)*SIZE,hipMemcpyHostToDevice);
	
	input<<<256,512>>>(carr2,carr);
	hipMemcpy(arr2,carr2,sizeof(int)*SIZE,hipMemcpyDeviceToHost);

	
	hipFree(carr2);
	hipFree(carr);
	free(arr2);
	free(arr);

	return 0;
}
