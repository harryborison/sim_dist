
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>
#define BUF_SIZ 1000000
__global__ void addKernel(int *c , const int *a, const int *b){
	int i = threadIdx.x;
	c[i] = a[i]+ b[i];
}
__global__ void GetKernel(int *b)
{
	int i = threadIdx.x;
	b[i] = b[i] * 10;


}
int main(void)
{
	int a[BUF_SIZ];
	int b[BUF_SIZ];
	int *ary1=0;
//	int *ary2=0;
//	int *ary3=0;
	for(int i=0;i<BUF_SIZ;i++)
	{
		a[i] = i;
	}
	hipMalloc((void**)&ary1 , BUF_SIZ*sizeof(int));
//	cudaMalloc((void**)&ary2 , BUF_SIZ*sizeof(int));
//	cudaMalloc((void**)&ary3 , BUF_SIZ*sizeof(int));
	
//	cudaMemcpy(ary2, a, BUF_SIZ*sizeof(int),cudaMemcpyHostToDevice);
	hipMemcpy(ary1, a, BUF_SIZ*sizeof(int),hipMemcpyHostToDevice);
	/*

	for(int i=0;i<100000;i++)
	{
		ary3[i] = ary1[i] + ary2[i];
	}
	*/
	printf("addkernel start\n");
//	addKernel<<<1,3>>>(ary3,ary1,ary2);
	GetKernel<<<1,3>>>(ary1);	
	printf("addkernel end\n");
//	cudaMemcpy(b, ary3 ,BUF_SIZ*sizeof(int),cudaMemcpyDeviceToHost);
	hipMemcpy(b, ary1 ,BUF_SIZ*sizeof(int),hipMemcpyDeviceToHost);

	for(int i =0; i < BUF_SIZ; i++)
	{
		if(i/500 ==0)
		{
			printf("%d ",b[i]);
		}
	}
	hipFree(ary1);
//	cudaFree(ary2);
//      cudaFree(ary3);
	return 0;


}
