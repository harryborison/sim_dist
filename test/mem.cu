
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>

int main(void)
{
	int a[100000];
	int b[100000];
	int *ary1;
	for(int i=0;i<100000;i++)
	{
		a[i] = i;
	}
	hipMalloc((void**)&ary1 , 100000*sizeof(int));
	hipMemcpy(ary1, a, 100000*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(b, ary1 ,100000*sizeof(int),hipMemcpyDeviceToHost);
	for(int i =0; i < 100000; i++)
	{
		if(i/500 ==0)
		{
			printf("%d ",i);
		}
	}
	hipFree(ary1);
	return 0;


}
