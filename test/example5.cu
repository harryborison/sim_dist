//example5.cu


#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>
__global__ void addKernel(int *c , const int *a, const int *b){
	int i = threadIdx.x;
	c[i] = a[i]+ b[i];
}
int main(void)
{
	int a[200000];
	int b[200000];
	int *ary1=0;
	int *ary2=0;
	int *ary3=0;
	for(int i=0;i<200000;i++)
	{
		a[i] = i;
	}
	hipMalloc((void**)&ary1 , 200000*sizeof(int));
	hipMalloc((void**)&ary2 , 200000*sizeof(int));
	hipMalloc((void**)&ary3 , 200000*sizeof(int));
	
	hipMemcpy(ary2, a, 200000*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(ary1, a, 200000*sizeof(int),hipMemcpyHostToDevice);
	/*

	for(int i=0;i<100000;i++)
	{
		ary3[i] = ary1[i] + ary2[i];
	}
	*/
	addKernel<<<1,3>>>(ary3,ary1,ary2);
	hipMemcpy(b, ary3 ,200000*sizeof(int),hipMemcpyDeviceToHost);
	for(int i =0; i < 200000; i++)
	{
		if(i/500 ==0)
		{
			printf("%d ",b[i]);
		}
	}
	hipFree(ary1);
	hipFree(ary2);
	hipFree(ary3);
	return 0;


}
