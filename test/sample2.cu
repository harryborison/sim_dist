
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>

#define SIZE 256*1024*64

__global__ void input(int *a, int *b)
{
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	a[i]=b[i];
}


int main(void)
{
	int *arr;
	int *arr2;
	int *carr=0;
	int *carr2=0;
	arr= (int *)malloc(sizeof(int)*SIZE);
	arr2= (int *)malloc(sizeof(int)*SIZE);
	for(int i=0; i<SIZE; i++)
	{
	arr[i] = 2;

	}
	printf("%d %d",arr[SIZE-1],arr[SIZE-100]);	

	hipMalloc((void**)&carr2,sizeof(int)*SIZE);
	hipMalloc((void**)&carr,sizeof(int)*SIZE);
	hipMemcpy(carr,arr,sizeof(int)*SIZE,hipMemcpyHostToDevice);
	
	input<<<9096,512>>>(carr2,carr);
	hipMemcpy(arr2,carr2,sizeof(int)*SIZE,hipMemcpyDeviceToHost);

	printf("output : %d %d %d",arr2[0],arr2[10000],arr2[1000]);
	
	hipFree(carr2);
	hipFree(carr);
	free(arr2);
	free(arr);

	return 0;
}
