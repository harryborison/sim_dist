
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <stdlib.h>
#define WIDTH 10000 
typedef struct input{
	int x;
}input;
__global__ void inputkernel(input *c, const input *a)
{
	int i = threadIdx.x + blockIdx.x *100; 
	c[i].x = a[i].x+1;	
}


int main(void)
{

	input *inputt=0;
	input *minput=0;
	int *cary=0;
	int *cary2=0;
	int *ary;
	minput = (input *)malloc(sizeof(input)*WIDTH);

	hipMalloc((void **)&inputt , WIDTH * sizeof(input));
			
	for(int i = 0 ; i < WIDTH ; i++)
	{
	minput[i].x=0;	


	}
	

	inputkernel<<<10,100>>>(inputt,minput);

	
	printf("kernel exit\n");
	hipMemcpy(minput,inputt,sizeof(input)*WIDTH,hipMemcpyDeviceToHost);

	printf("final result : %d %d %d", minput[0].x,minput[1].x ,minput[9999].x );
	hipFree(inputt);
	return 0;


}

